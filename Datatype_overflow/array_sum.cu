#include "hip/hip_runtime.h"
/*
 *
 * Datatype overflow from GPU
 * array sum
 *
 */
 
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "device_launch_parameters"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ 
void array_sum(int n, int *e_x, int *e_z) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
	//Inserted Error: e_z out of bounds
	for (id = 0; id < n; id++) {
		*e_z += e_x[id];
	}
}

int main() {
    int no_el = 1048576;
    int block_size = 512;
	int grid_size = (no_el/block_size) + 1;                    //ceil doesn't give correct grid size
	
	int *h_x, *d_x, *h_z, *d_z;
	 
	h_x = (int*)malloc(no_el*sizeof(int));
	h_z = (int*)malloc(sizeof(int));

    *h_z = 0;
    for (int i = 0; i < no_el; i++) {
	    h_x[i] = i;												//sum = 524288(0+1048575)=already out of bounds
	}
	
    hipMalloc(&d_x, no_el*sizeof(int)); 
	hipMalloc(&d_z, sizeof(int));
	
	hipMemcpy(d_x, h_x, no_el*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_z, h_z, sizeof(int), hipMemcpyHostToDevice);
	
	dim3 block(block_size);
	dim3 grid(grid_size);
	
	array_sum<<<grid, block>>>(no_el, d_x, d_z);
	
	gpuErrchk(hipMemcpy(h_z, d_z, sizeof(int), hipMemcpyDeviceToHost));
	
	hipFree(d_x);
    hipFree(d_z);
 
    free(h_x);
    free(h_z);
	
}	 