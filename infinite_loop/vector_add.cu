/*
 *
 * Accessing out of bound memory from GPU
 * Vector addition
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__
void vector_addition(int n, int *e_x, int *e_y, int *e_z) {
    //int id = blockIdx.x * blockDim.x + threadIdx.x;
    //Inserted Error: while(1) loop
    while(1) {
    }
}

int main() {
    int no_el = 1048576;
    int block_size = 512;
    int grid_size = (no_el/block_size) + 1;                    //ceil doesn't give correct grid size

    int *h_x, *d_x, *h_y, *d_y, *h_z, *d_z;

    h_x = (int*)malloc(no_el*sizeof(int));
    h_y = (int*)malloc(no_el*sizeof(int));
    h_z = (int*)malloc(no_el*sizeof(int));

    for (int i = 0; i < no_el; i++) {
        h_x[i] = i;
        h_y[i] = i + 2;
        h_z[i] = 0;
    }

    hipMalloc(&d_x, no_el*sizeof(int));
    hipMalloc(&d_y, no_el*sizeof(int));
    hipMalloc(&d_z, no_el*sizeof(int));

    hipMemcpy(d_x, h_x, no_el*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, no_el*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_z, h_z, no_el*sizeof(int), hipMemcpyHostToDevice);

    dim3 block(block_size);
    dim3 grid(grid_size);

    vector_addition<<<grid, block>>>(no_el, d_x, d_y, d_z);

    gpuErrchk(hipMemcpy(h_z, d_z, no_el*sizeof(int), hipMemcpyDeviceToHost));

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);

    free(h_x);
    free(h_y);
    free(h_z);
}
