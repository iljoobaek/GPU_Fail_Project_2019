#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

// device memory
float* dev_A;
float* dev_B;
float* dev_out;

// host memory
float* matrix1;
float* matrix2;
float* outBuffer;


inline void CHECKCUDA(hipError_t e)
{
	if (e != hipSuccess)
	{
		cerr<<"CUDA Error:"<< hipGetErrorString(e) << endl;
		exit(1);
	}
}

__device__
void MatAddCuda(float* A, float* B, float* out, int idx)
{
    out[idx] = A[idx]+B[idx];
}


__global__
void MatCalculate(float *A, float *B, float*out, int iteration_cnt)
{
    int idx = threadIdx.x;
    int i=0, j=0;
    //for (i = 0; i < ITERATION_NUM*2; i++) {
    for (i = 0; i < iteration_cnt; i++) {
        for (j= 0; j < iteration_cnt; j++)
            MatAddCuda(A, B, out, idx);
    }
}


void cudaTestWrapper_MatCalculate(float* A, float* B, int size, int calType, float* out, int iteration_cnt)
{
    CHECKCUDA(hipMemcpy(dev_A, A, size* sizeof(float), hipMemcpyHostToDevice));
    CHECKCUDA(hipMemcpy(dev_B, B, size* sizeof(float), hipMemcpyHostToDevice));

    dim3 numBlocks(1);
    dim3 threadsPerBlock(size);

    switch(calType)
    {
        case 1: // sum
            //cout << "cuda sum " << size << endl;
            MatCalculate<<<numBlocks, threadsPerBlock, 0>>>(dev_A,dev_B,dev_out, iteration_cnt);
            break;
        case 2: // multiplication
            break;
        default :
            break;
    }
    hipDeviceSynchronize();

    CHECKCUDA(hipMemcpy(out, dev_out, size * sizeof(float), hipMemcpyDeviceToHost));
}


void cudaInit(int size)
{
    // Copying structure hue and sat to cudaMemCpy
    CHECKCUDA(hipMalloc(&dev_A, size * sizeof(float)));
    CHECKCUDA(hipMalloc(&dev_B, size * sizeof(float)));
    CHECKCUDA(hipMalloc(&dev_out, size * sizeof(float)));

    CHECKCUDA(hipHostMalloc((float **) &matrix1, size* sizeof(float)));
    CHECKCUDA(hipHostMalloc((float **) &matrix2, size* sizeof(float)));
    CHECKCUDA(hipHostMalloc((float **) &outBuffer, size* sizeof(float)));
}

void cudaExit()
{
    //cout << "cudaExit" << endl;
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_out);

    hipHostFree(matrix1);
    hipHostFree(matrix2);
    hipHostFree(outBuffer);
}
